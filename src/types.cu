#include "hip/hip_runtime.h"
#include "types.cuh"
#include "utils.cuh"

#include <numeric>
#include <functional>
#include <cassert>

#include <iostream>

#define DEF_CLASS_SPEC(name)\
    template class name<Npp8u>;\
    template class name<Npp8s>;\
    template class name<Npp16u>;\
    template class name<Npp16s>;\
    template class name<Npp32u>;\
    template class name<Npp32s>;\
    template class name<Npp64u>;\
    template class name<Npp64s>;\
    template class name<Npp32f>;\
    template class name<Npp64f>;

namespace cas
{

/////////////////////////////////////////////////////
/// SAFE STREAM
////////////////////////////////////////////////////

///
/// \brief safe_stream : default constructor.
/// Initialize the attribute "stream" to nullptr.
///
__host__ safe_stream::safe_stream():
    stream(nullptr)
{}

///
/// \brief safe_stream : parametric constructor.
/// \param _stream : stream to own.
/// The stream provided as argument is own
/// by the object, and will be destroy either
/// by calling the method "destroy" of by
/// the destructor.
///
__host__ safe_stream::safe_stream(const hipStream_t& _stream):
    stream(_stream)
{}

///
/// \brief ~safe_stream : destructor
/// If a stream was created, it will be
/// destroy.
///
__host__ safe_stream::~safe_stream()
{
    this->destroy();

}

///
/// \brief create : create a new stream.
/// \param flags : attributes of the stream to set.
/// \param priority : priority to set for the stream.
///
__host__ void safe_stream::create(const unsigned int& flags, const int& priority)
{
    // If a stream already exists
    // it must be destroied before
    // continuing.
    if(this->stream)
        this->destroy();

    if(flags == hipStreamDefault && priority<0)
    {
        check_cuda_error_or_npp_status(hipStreamCreate(std::addressof(this->stream)));
    }
    else if(flags != hipStreamDefault && priority<0)
    {
        check_cuda_error_or_npp_status(hipStreamCreateWithFlags(std::addressof(this->stream), flags));
    }
    else if(flags == hipStreamDefault && priority>=0)
    {
        check_cuda_error_or_npp_status(hipStreamCreateWithPriority(std::addressof(this->stream), hipStreamDefault, priority));
    }
    else
    {
        check_cuda_error_or_npp_status(hipStreamCreateWithPriority(std::addressof(this->stream), flags, priority));
    }
}

///
/// \brief destroy : destroy the current
/// stream, if it was created. Otherwise
/// do nothing.
///
__host__ void safe_stream::destroy()
{
    if(this->stream)
    {
        check_cuda_error_or_npp_status(hipStreamDestroy(this->stream));
        this->stream = nullptr;
    }
}

///
/// \brief waitEvent : wait for an event to finish.
/// \param event : event to monitor.
/// \param flags : attributes to set.
///
__host__ safe_stream::operator hipStream_t() const
{
    return this->stream;
}

///
/// \brief operator hipStream_t : implicit conversion operator.
/// Convinient to maintain compatibility with the rest of the
/// CUDA API.
///
__host__ void safe_stream::waitEvent(const safe_event& event, const unsigned int& flags)
{
    check_cuda_error_or_npp_status(hipStreamWaitEvent(this->stream, event, flags));
}


/////////////////////////////////////////////////////
/// SAFE EVENT
////////////////////////////////////////////////////

///
/// \brief safe_event : parametric constructor.
/// \param event : event to own.
/// The stream provided as argument is own
/// by the object, and will be destroy either
/// by calling the method "destroy" of by
/// the destructor.
///
__host__ safe_event::safe_event():
    event(nullptr)
{}


///
/// \brief ~safe_event : destructor
/// If an event was created, it will be
/// destroy.
///
__host__ safe_event::~safe_event()
{
    if(this->event)
        this->destroy();
}

///
/// \brief create : create the event.
/// \param flags : attriutes to apply on the event.
///
__host__ void safe_event::create(const unsigned int& flags)
{
    if(this->event)
        this->destroy();

    if(flags!=hipEventDefault)
    {
        check_cuda_error_or_npp_status(hipEventCreate(std::addressof(this->event)));
    }
    else
    {
        check_cuda_error_or_npp_status(hipEventCreateWithFlags(std::addressof(this->event), flags));
    }
}

///
/// \brief destroy : destroy the current
/// event, if it was created. Otherwise
/// do nothing.
///
__host__ void safe_event::destroy()
{
    if(this->event)
    {
        check_cuda_error_or_npp_status(hipEventDestroy(this->event));
        this->event = nullptr;
    }
}

///
/// \brief record : record an event
/// \param _stream : stream to record.
/// \param _flags : attributes to apply on the recording.
///
__host__ void safe_event::record(const safe_stream& _stream, const unsigned int& _flags)
{
    if(_flags!=hipEventDefault)
    {
        check_cuda_error_or_npp_status(hipEventRecord(this->event, _stream));
    }
    else
    {
        check_cuda_error_or_npp_status(hipEventRecordWithFlags(this->event, _stream, _flags));
    }
}

///
/// \brief synchonize : waits for an event to complete.
///
__host__ void safe_event::synchonize()
{
    check_cuda_error_or_npp_status(hipEventSynchronize(this->event));
}

///
/// \brief operator hipEvent_t : implicit conversion operator.
/// Convinient to maintain compatibility with the rest of the
/// CUDA API.
///
__host__ safe_event::operator hipEvent_t() const
{
    return this->event;
}


/////////////////////////////////////////////////////
/// SIMPLE VECTOR
////////////////////////////////////////////////////

///
/// \brief nppiVector_t : default constructor.
/// Initialize the data pointer and the counter
/// to null, and the dimensionality attributes to 0
///
template<class T>
__host__ nppiVector_t<T>::nppiVector_t():
    data(nullptr),
    len(0)
{}

///
/// \brief nppiVector_t
/// \param size
///
template<class T>
__host__ nppiVector_t<T>::nppiVector_t(const Npp32s& size):
    nppiVector_t()
{
    this->create(size);
}

///
/// \brief nppiVector_t :
/// \param _data
/// \param size
/// \param own
///
template<class T>
__host__ nppiVector_t<T>::nppiVector_t(pointer _data, const Npp32s size, const bool& own):
    data(_data),
    len(size),
    counter(own ? new int(1) : nullptr)
{}

///
/// \brief nppiVector_t
/// \param obj
///
template<class T>
__host__ nppiVector_t<T>::nppiVector_t(const nppiVector_t& obj):
    data(obj.data),
    len(obj.len),
    counter(obj.counter)
{
    if(this->counter)
        ++(*this->counter);
}


///
/// \brief ~nppiVector_t
///
template<class T>
__host__ nppiVector_t<T>::~nppiVector_t()
{
    this->release();
}

///
/// \brief operator =
/// \param obj
/// \return
///
template<class T>
__host__ nppiVector_t<T>& nppiVector_t<T>::operator=(const nppiVector_t& obj)
{

    if(this != std::addressof(obj))
    {
        this->release();

        this->data = obj.data;
        this->len = obj.len;
        this->counter = obj.counter;

        if(this->counter)
            ++(*this->counter);
    }

    return (*this);
}



///
/// \brief size
/// \return
///
template<class T>
__host__ Npp32s nppiVector_t<T>::size() const
{
    return this->len;
}

///
/// \brief ptr
/// \param i
/// \return
///
template<class T>
__host__ typename nppiVector_t<T>::pointer nppiVector_t<T>::ptr(const Npp32s& i)
{
    return reinterpret_cast<pointer>(this->data + i * sizeof(value_type));
}

///
/// \brief ptr
/// \param i
/// \return
///
template<class T>
__host__ typename nppiVector_t<T>::const_pointer nppiVector_t<T>::ptr(const Npp32s& i)const
{
    return reinterpret_cast<const_pointer>(this->data + i * sizeof(value_type));
}

///
/// \brief create : memory allocation method
/// \param size : size to allocate or reallocate.
///
template<class T>
__host__ void nppiVector_t<T>::create(const Npp32s& _size)
{
    if(this->len != _size)
    {
        this->release();

        this->len = _size;

        this->data = nppsMalloc_8u(this->len * sizeof(value_type));
    }
}

///
/// \brief release : memory release method.
/// If the memory is own and the counter
/// after decrementation has reach 0,
/// then the memory is deallocated.
/// In any cases the attrobutes are reset
/// to null for the address and the counter
/// and 0 for the dimensionality attributes.
///
template<class T>
__host__ void nppiVector_t<T>::release()
{
    if(this->data && this->counter && !(--(*this->counter)) )
        nppsFree(this->data);
    this->data = nullptr;
    this->len = 0;
    this->counter.reset();
}


/////////////////////////////////////////////////////
/// SIMPLE MATRIX
////////////////////////////////////////////////////

///
/// \brief nppiMatrix_t : default constructor.
/// Initialize the data pointer and the counter
/// to null, and the dimensionality attributes to 0
///
template<class T>
__host__ nppiMatrix_t<T>::nppiMatrix_t():
    data(nullptr),
    rows(0),
    cols(0),
    step(0)
{}

///
/// \brief nppiMatrix_t : parametrict constructor.
/// Allocate memory in order to at least host a matrix
/// which dimensions are specify by the inputs.
/// \param _rows : number of rows of the matrix to create.
/// \param _cols : number of colmuns of the matrix to create.
///
template<class T>
__host__ nppiMatrix_t<T>::nppiMatrix_t(const Npp32s& _rows, const Npp32s& _cols)
{
    this->create(_rows, _cols);
}

///
/// \brief nppiMatrix_t : parametric constructor.
/// This constructor is an interface with memory
/// allocation outside of the class. If memory
/// can be own it will be deallocated by the current
/// object, otherwise it will not be deallocate
/// by the current object.
/// \param data :
/// \param _step
/// \param _rows
/// \param _cols
/// \param _own
///
template<class T>
__host__ nppiMatrix_t<T>::nppiMatrix_t(pointer _data, const Npp32s& _step, const Npp32s& _rows, const Npp32s& _cols, const bool& _own):
    data(_data),
    rows(_rows),
    cols(_cols),
    step(_step),
    counter(_own ? new int(1) : nullptr)
{}

///
/// \brief nppiMatrix_t : copy constructor,
/// Initialize the current object to the same
/// values as those of the provided object.
/// If the counter is initialize, it is incremented.
/// This constructor DOES NOT perform any copy.
/// \param obj : object to initialize the attributes on.
///
template<class T>
__host__ nppiMatrix_t<T>::nppiMatrix_t(const nppiMatrix_t &obj):
    data(obj.data),
    rows(obj.rows),
    cols(obj.cols),
    step(obj.step),
    counter(obj.counter)
{}


///
/// \brief ~nppiMatrix_t : destructor.
/// If the memory is own and the counter
/// after decrementation has reach 0,
/// then the memory is deallocated.
/// In any cases the attrobutes are reset
/// to null for the address and the counter
/// and 0 for the dimensionality attributes.
///
template<class T>
__host__ nppiMatrix_t<T>::~nppiMatrix_t()
{
    this->release();
}


///
/// \brief Assignation operator :
/// Initialize the current object to the same
/// values as those of the provided object.
/// If the counter is initialize, it is incremented.
/// This operator DOES NOT perform any copy.
/// \param obj : object to initialize the attributes on.
/// \return current object.
///
template<class T>
__host__ nppiMatrix_t<T>& nppiMatrix_t<T>::operator=(const nppiMatrix_t& obj)
{
    if(std::addressof(obj) != this)
    {
        this->data = obj.data;
        this->rows = obj.rows;
        this->cols = obj.cols;
        this->step = obj.step;
        this->counter = obj.counter;

        if(this->counter)
            ++(*this->counter);
    }

    return (*this);
}

///
/// \brief release : memory release method.
/// If the memory is own and the counter
/// after decrementation has reach 0,
/// then the memory is deallocated.
/// In any cases the attrobutes are reset
/// to null for the address and the counter
/// and 0 for the dimensionality attributes.
///
template<class T>
__host__ void nppiMatrix_t<T>::release()
{
    if(this->counter && this->counter && !(--(*this->counter)))
        nppiFree(this->data);

    this->data = nullptr;
    this->rows = this->cols = this->step = 0;
    this->counter.reset();
}


///
/// \brief create : memory allocation method.
/// Allocate memory in order to at least host a matrix
/// which dimensions are specify by the inputs.
/// \param _rows : number of rows of the matrix to create.
/// \param _cols : number of colmuns of the matrix to create.
///
template<class T>
__host__ void nppiMatrix_t<T>::create(const Npp32s& _rows, const Npp32s& _cols)
{
    if(_rows != this->rows || _cols != this->cols)
        this->release();

    this->rows = _rows;
    this->cols = _cols;
    this->data = reinterpret_cast<T*>(nppiMalloc_8u_C1(this->cols * sizeof(T), this->rows, &this->step));
    this->counter.reset(new int (1));
}

///
/// \brief size : accessor.
/// Return a NppiSize object.
/// \return the width and height of the current matrix
/// allocation.
///
template<class T>
__host__ NppiSize nppiMatrix_t<T>::size() const
{
    return {this->cols, this->rows};
}


///
/// \brief width : accessor.
/// \return the width of the current matrix.
///
template<class T>
__host__ Npp32s nppiMatrix_t<T>::width() const
{
    return this->cols;
}


///
/// \brief height : accessor.
/// \return the height of the current matrix.
///
template<class T>
__host__ Npp32s nppiMatrix_t<T>::height() const
{
    return this->rows;
}


///
/// \brief pitch : accessor.
/// Return the line step of the current memory allocation
/// \note The line step ensure that the memory is aligned.
/// \return the line step of the current matrix.
///
template<class T>
__host__ Npp32s nppiMatrix_t<T>::pitch() const
{
    return this->step;
}


///
/// \brief ptr : accessor.
/// return the address of the first element of the specified row.
/// \param y : index of the row to return the address of the first element of.
/// \return address of the first element of the specified specified by the row.
///
template<class T>
__host__ typename nppiMatrix_t<T>::pointer nppiMatrix_t<T>::ptr(const Npp32s& y)
{
    return reinterpret_cast<pointer>(reinterpret_cast<unsigned char*>(this->data) + y * this->step);
}


///
/// \brief ptr : accessor.
/// return the address of the first element of the specified row.
/// \param y : index of the row to return the address of the first element of.
/// \return address of the first element of the specified specified by the row.
///
template<class T>
__host__ typename nppiMatrix_t<T>::const_pointer nppiMatrix_t<T>::ptr(const Npp32s& y) const
{
    return reinterpret_cast<const_pointer>(reinterpret_cast<const unsigned char*>(this->data) + y * this->step);
}


///
/// \brief ptr : accessor.
/// return the address of the element of the specified row and column.
/// \param y : index of the row.
/// \param x : index of the columns.
/// \return address of the element located that the y^{th} rows and x^{th} rows..
///
template<class T>
__host__ typename nppiMatrix_t<T>::pointer nppiMatrix_t<T>::ptr(const Npp32s& y, const Npp32s& x)
{
    return this->ptr(y) + x;
}


///
/// \brief ptr : accessor.
/// return the address of the element of the specified row and column.
/// \param y : index of the row.
/// \param x : index of the columns.
/// \return address of the element located that the y^{th} rows and x^{th} rows..
///
template<class T>
__host__ typename nppiMatrix_t<T>::const_pointer nppiMatrix_t<T>::ptr(const Npp32s& y, const Npp32s& x) const
{
    return this->ptr(y) + x;
}

DEF_CLASS_SPEC(nppiMatrix_t)



/////////////////////////////////////////////////////
/// SIMPLE TENSOR
////////////////////////////////////////////////////


///
/// \brief nppiTensor_t : parametric constructor.
/// This constructor is an interface with memory
/// allocation outside of the class. If memory
/// can be own it will be deallocated by the current
/// object, otherwise it will not be deallocate
/// by the current object.
/// \param data :
/// \param _step
/// \param _rows
/// \param _cols
/// \param _own
///
template<class T>
__host__ nppiTensor_t<T>::nppiTensor_t(pointer _data, const std::vector<Npp32s>& _steps, const std::vector<Npp32s>& _dimensions, const bool& _own):
    data(reinterpret_cast<unsigned char*>(_data)),
    steps(_steps),
    dims(_dimensions),
    counter(_own ? new int(1) : nullptr)
{}

///
/// \brief nppiTensor_t : copy constructor,
/// Initialize the current object to the same
/// values as those of the provided object.
/// If the counter is initialize, it is incremented.
/// This constructor DOES NOT perform any copy.
/// \param obj : object to initialize the attributes on.
///
template<class T>
__host__ nppiTensor_t<T>::nppiTensor_t(const nppiTensor_t& obj):
    data(obj.data),
    steps(obj.steps),
    dims(obj.dims),
    counter(obj.counter)
{
    if(this->counter)
        ++(*this->counter);
}



///
/// \brief ~nppiTensor_t : destructor.
/// If the memory is own and the counter
/// after decrementation has reach 0,
/// then the memory is deallocated.
/// In any cases the attrobutes are reset
/// to null for the address and the counter
/// and 0 for the dimensionality attributes.
///
template<class T>
__host__ nppiTensor_t<T>::~nppiTensor_t()
{
    this->release();
}

///
/// \brief Assignation operator :
/// Initialize the current object to the same
/// values as those of the provided object.
/// If the counter is initialize, it is incremented.
/// This operator DOES NOT perform any copy.
/// \param obj : object to initialize the attributes on.
/// \return current object.
///
template<class T>
__host__ nppiTensor_t<T>& nppiTensor_t<T>::operator=(const nppiTensor_t& obj)
{

    if(std::addressof(obj) != this)
    {
        this->release();

        this->data = obj.data;
        this->dims = obj.dims;
        this->steps = obj.steps;
        this->counter = obj.counter;

        if(this->counter)
            ++(*this->counter);
    }

    return (*this);

}



///
/// \brief order :
/// return the tensor order. (0 if it is a scalar, 1 for a vector, ...)
/// \return order of the current tensor.
///
template<class T>
__host__ Npp32s nppiTensor_t<T>::order() const
{
    return !this->dims.front() ? 0 : static_cast<Npp32s>(this->steps.size());
}


///
/// \brief dimensions :
/// return the dimensions of the current array
/// \return
///
template<class T>
__host__ std::vector<Npp32s> nppiTensor_t<T>::dimensions() const
{
    return this->dims;
}

///
/// \brief dimension :
/// return the value of the specified dimension.
/// \param idx : dimension to know about.
/// \return value of the specified dimension.
///
template<class T>
__host__ Npp32s nppiTensor_t<T>::dimension(const Npp32s& idx) const
{
    assert(idx<static_cast<Npp32s>(this->dims.size()));
    return this->dims.at(idx);
}

///
/// \brief pitchs :
/// return the value of the pitchs for all the dimensions as number of bytes.
/// \return value of the pitchs for all the dimensions as number of bytes.
///
template<class T>
__host__ std::vector<Npp32s> nppiTensor_t<T>::pitchs() const
{
    std::vector<Npp32s> ret = this->steps;

    for(Npp32s& step : ret)
        step *= sizeof(T);

    return ret;
}

///
/// \brief pitch :
/// return the value of pitch for the specified dimension as number of bytes.
/// \param idx : dimension to know about.
/// \return value of pitch for the specified dimension as number of bytes.
///
template<class T>
__host__ Npp32s nppiTensor_t<T>::pitch(const Npp32s& idx) const
{
    return this->steps.at(idx) * sizeof(T);
}

///
/// \brief release : memory release method.
/// If the memory is own and the counter
/// after decrementation has reach 0,
/// then the memory is deallocated.
/// In any cases the attrobutes are reset
/// to null for the address and the counter
/// and 0 for the dimensionality attributes.
///
template<class T>
__host__ void nppiTensor_t<T>::release()
{
    if(this->counter && this->counter && !(--(*this->counter)))
        nppiFree(this->data);

    this->data = nullptr;
    this->dims.clear();
    this->steps.clear();
    this->counter.reset();
}


template<class T>
__host__ void nppiTensor_t<T>::create(const std::vector<int>& dimensions)
{

    // Prepare the dimensions attribute.
    this->dims = std::move(dimensions);

    Npp32s step0 = std::accumulate(this->dims.begin(), this->dims.end(), 1, std::multiplies<Npp32s>()) * sizeof(value_type);

    this->data = nppsMalloc_8u(step0);

    // Step 0 correspond to the product of all the dimensions starting from the second.
    // e.g. type: float32 (i.e. sizeof -> 4),  dims : 32 x 3 x 640 x 480 -> steps: (3 x 640 x 480 x sizeof(float32)) x (640 x 480 x sizeof(float32)) x (480 x sizeof(float32)) x sizeof(float32)
    this->steps.push_back(step0);

    for(size_t i=1;i<this->dims.size(); ++i)
        this->steps.push_back(std::accumulate(this->dims.begin() + i, this->dims.end(), 1, std::multiplies<Npp32s>()) * sizeof(value_type));

    // One might observe or know that step0 value might be larger than product of all, but the first, dimensions.
    // From a practical point of view this will results in some bytes not used. This slight over-allocation allocation
    // ensure that the memory is always aligned.

}



///
/// \brief ptr : accessor.
/// return the address of the element of the specified row and column.
/// \param y : index of the first element on the dimension.
/// \param indices : index of the elements of all the other dimensions, but the first.
/// \return address of the element located that the y^{th} rows and x^{th} rows..
///
template<class T>
__host__ typename nppiTensor_t<T>::pointer nppiTensor_t<T>::ptr(const std::initializer_list<int>& indices)
{
    return reinterpret_cast<pointer>(this->data + this->get_index(indices));
}

///
/// \brief ptr : accessor.
/// return the address of the element of the specified row and column.
/// \param y : index of the first element on the dimension.
/// \param indices : index of the elements of all the other dimensions, but the first.
/// \return address of the element located that the y^{th} rows and x^{th} rows..
///
template<class T>
__host__ typename nppiTensor_t<T>::const_pointer nppiTensor_t<T>::ptr(const std::initializer_list<int>& indices) const
{
    return reinterpret_cast<const_pointer>(this->data + this->get_index(indices));
}


///
/// \brief get_index :
/// compute the index of the pointer
/// corresponding to a set of coordinates.
/// \param indices
/// \return address of the element corresponding to the provided coordinates.
///
template<class T>
Npp32s nppiTensor_t<T>::get_index(const std::initializer_list<int>& _indices)const
{
    std::vector<int> indices = _indices;

    Npp32s ret(0);

    for(size_t i=0; i<std::min(this->steps.size(), indices.size()); ++i)
        ret += indices.at(i) * this->steps.at(i);

    return ret;
}


DEF_CLASS_SPEC(nppiTensor_t)

} // cas
